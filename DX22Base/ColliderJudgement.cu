//	ColliderJudgement.cu

#include "ColliderJudgement.cuh"

__host__ ColliderJudgement::ColliderJudgement()
{
	m_CubeFrame.push_back({ -0.5f, 0.5f,-0.5f });//0
	m_CubeFrame.push_back({  0.5f, 0.5f,-0.5f });//1
	m_CubeFrame.push_back({ -0.5f,-0.5f,-0.5f });//2
	m_CubeFrame.push_back({  0.5f,-0.5f,-0.5f });//3
	m_CubeFrame.push_back({ -0.5f, 0.5f, 0.5f });//4
	m_CubeFrame.push_back({  0.5f, 0.5f, 0.5f });//5
	m_CubeFrame.push_back({ -0.5f,-0.5f, 0.5f });//6
	m_CubeFrame.push_back({  0.5f,-0.5f, 0.5f });//7
}

__host__ ColliderJudgement::~ColliderJudgement()
{
}

__host__ void ColliderJudgement::HitJudgement()
{
	char* D_Ans[JudgeTag::JUDGE_TAGMAX];
	
	dim3 block, grid;//�O���b�h���Ƀu���b�N

}